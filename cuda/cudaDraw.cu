#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaDraw.h"
#include "cudaAlphaBlend.cuh"


// TODO for rect/fill/line
//    - make versions that only accept image (as both input/output)
//    - add line width/line color
//    - add overloads for single shape/multiple shapes
//    - benchmarking of copy vs alternate kernel when input != output
//    - overloads using int2 for coordinates
//    - add a template parameter for alpha blending

#define MIN(a,b)  (a < b ? a : b)
#define MAX(a,b)  (a > b ? a : b)

template<typename T> inline __device__ __host__ T sqr(T x) 				    { return x*x; }

inline __device__ __host__ float dist2(float x1, float y1, float x2, float y2) { return sqr(x1-x2) + sqr(y1-y2); }
inline __device__ __host__ float dist(float x1, float y1, float x2, float y2)  { return sqrtf(dist2(x1,y1,x2,y2)); }


//----------------------------------------------------------------------------
// Circle drawing (find if the distance to the circle <= radius)
//----------------------------------------------------------------------------						 
template<typename T>
__global__ void gpuDrawCircle( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int cx, int cy, float radius2, const float4 color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int dx = x - cx;
	const int dy = y - cy;
	
	// if x,y is in the circle draw it
	if( dx * dx + dy * dy < radius2 ) 
	{
		const int idx = y * imgWidth + x;
		img[idx] = cudaAlphaBlend(img[idx], color);
	}
}


template<typename T>
__global__ void gpuDrawCircleOnY( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int cx, int cy, float radius2, const float4 color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int dx = x - cx;
	const int dy = y - cy;
	
	// if x,y is in the circle draw it
	if( dx * dx + dy * dy < radius2 ) 
	{
		// const int idx = y * imgWidth + x;
		const int idx = /* (char *)pDevPtr  + */ y * 2048 + x;
		img[idx] = 255;
	}
}

template<typename T>
__global__ void gpuDrawCircleOnOnePlane( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int cx, int cy, float radius2, uint8_t color, int pitch ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int dx = x - cx;
	const int dy = y - cy;
	
	// if x,y is in the circle draw it
	if( dx * dx + dy * dy < radius2 ) 
	{
		// const int idx = y * imgWidth + x;
		const int idx = /* (char *)pDevPtr  + */ y * pitch + x;
		img[idx] = color;
	}
}

/**
 * @brief this for drawwing circle on yuyv image in y planes...
 * 
 * @param img 
 * @param imgWidth 
 * @param imgHeight 
 * @param offset_x 
 * @param offset_y 
 * @param cx 
 * @param cy 
 * @param radius2 
 * @param color 
 * @return __global__ 
 */
__global__ void gpuDrawCircleYY( unsigned char *img, int imgWidth, int imgHeight, int offset_x, int offset_y, int cx, int cy, float radius2, uint8_t color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int dx = x - cx;
	const int dy = y - cy;
	
	// if x,y is in the circle draw it
	if( dx * dx + dy * dy < radius2 ) 
	{
		const int idx = 2*y * imgWidth + 2*x;
		// const int idx = /* (char *)pDevPtr  + */ y * imgWidth + x;
		img[idx] = color;
	}
}

__global__ void gpuDrawCircleUV( unsigned char *img, int imgWidth, int imgHeight, int offset_x, int offset_y, int cx, int cy, float radius2, uint8_t color_u,uint8_t color_v ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int dx = x - cx;
	const int dy = y - cy;
	
	// if x,y is in the circle draw it
	if( dx * dx + dy * dy < radius2 ) 
	{
		const int idx_u = 4*y * imgWidth + 4*x+1;
		const int idx_v = 4*y * imgWidth + 4*x+3;
		// const int idx = /* (char *)pDevPtr  + */ y * imgWidth + x;
		img[idx_u] = color_u;
		img[idx_v] = color_v;
	}
}

inline __device__ void rgb_to_y(const uint8_t r, const uint8_t g, const uint8_t b, uint8_t& y)
{
	y = static_cast<uint8_t>(((int)(30 * r) + (int)(59 * g) + (int)(11 * b)) / 100);
}


inline __device__ void rgb_to_yuv(const uint8_t r, const uint8_t g, const uint8_t b, uint8_t& y, uint8_t& u, uint8_t& v)
{
	rgb_to_y(r, g, b, y);
	u = static_cast<uint8_t>(((int)(-17 * r) - (int)(33 * g) + (int)(50 * b) + 12800) / 100);
	v = static_cast<uint8_t>(((int)(50 * r) - (int)(42 * g) - (int)(8 * b) + 12800) / 100);
}

template<typename T>
__global__ void gpuDrawCircleOnYUV420( T* img_y,T* img_u,T* img_v, int imgWidth, int imgHeight, int offset_x, int offset_y, int cx, int cy, float radius2, const float4 color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int dx = x - cx;
	const int dy = y - cy;

	uint8_t y_val, u_val, v_val;
	rgb_to_yuv(uint8_t(color.x), uint8_t(color.y), uint8_t(color.z), y_val, u_val, v_val);
	
	// if x,y is in the circle draw it
	if( dx * dx + dy * dy < radius2 ) 
	{
		// const int idx = y * imgWidth + x;
		const int idx = y * 2048 + x;
		// const int idx_uv = y*512 + x/2;
		img_y[idx] = y_val;
		// if(x < imgWidth/8 && y <imgHeight/2)
		// {
			// img_u[idx_uv] = u_val;
			// img_v[idx_uv] = v_val;
		// }
	}

	// if( 2*dx * dx + 2*dy * dy < radius2 ) 
	// {
	// 	// const int idx = y * imgWidth + x;
	// 	// const int idx = y * 2048 + x;
	// 	const int idx_uv = y*512 + x/2;
	// 	// img_y[idx] = y_val;
	// 	// if(x < imgWidth/8 && y <imgHeight/2)
	// 	// {
	// 		img_u[idx_uv] = u_val;
	// 		img_v[idx_uv] = v_val;
	// 	// }
	// }
}



// cudaDrawCircle
hipError_t cudaDrawCircle( void* input, void* output, size_t width, size_t height, imageFormat format, int cx, int cy, float radius, const float4& color )
{
	if( !input || !output || width == 0 || height == 0 || radius <= 0 )
		return hipErrorInvalidValue;

	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the circle
	const int diameter = ceilf(radius * 2.0f);
	const int offset_x = cx - radius;
	const int offset_y = cy - radius;
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(diameter,blockDim.x), iDivUp(diameter,blockDim.y));

	#define LAUNCH_DRAW_CIRCLE(type) \
		gpuDrawCircle<type><<<gridDim, blockDim>>>((type*)output, width, height, offset_x, offset_y, cx, cy, radius*radius, color)
	
	if( format == IMAGE_RGB8 )
		LAUNCH_DRAW_CIRCLE(uchar3);
	else if( format == IMAGE_RGBA8 )
		LAUNCH_DRAW_CIRCLE(uchar4);
	else if( format == IMAGE_RGB32F )
		LAUNCH_DRAW_CIRCLE(float3); 
	else if( format == IMAGE_RGBA32F )
		LAUNCH_DRAW_CIRCLE(float4);
	else
	{
		imageFormatErrorMsg(LOG_CUDA, "cudaDrawCircle()", format);
		return hipErrorInvalidValue;
	}
		
	return hipGetLastError();
}

hipError_t cudaDrawCircleOnY( void* input, void* output, size_t width, size_t height, imageFormat format, int cx, int cy, float radius, const float4& color )
{
	// this is my function to draw cirle on Y channel of YUV image.. 
	if( !input || !output || width == 0 || height == 0 || radius <= 0 )
		return hipErrorInvalidValue;

	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the circle
	const int diameter = ceilf(radius * 2.0f);
	const int offset_x = cx - radius;
	const int offset_y = cy - radius;
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(diameter,blockDim.x), iDivUp(diameter,blockDim.y));

	#define LAUNCH_DRAW_CIRCLE_ON_Y(type) \
		gpuDrawCircleOnY<type><<<gridDim, blockDim>>>((type*)output, width, height, offset_x, offset_y, cx, cy, radius*radius, color)
	

	LAUNCH_DRAW_CIRCLE_ON_Y(uchar);		
	return hipGetLastError();
}

hipError_t  cudaDrawCircleOnYUYU( void* input, size_t width, size_t height, int cx, int cy, float radius, const float4& color )
{
	// this is my function to draw cirle on Y channel of YUV image.. 
	if( !input || width == 0 || height == 0 || radius <= 0 )
		return hipErrorInvalidValue;


	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	// if( input != output )
	// 	CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the circle

	const int diameter_y = ceilf(radius * 2.0f);
	const int offset_x = cx - radius;
	const int offset_y = cy - radius;
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(diameter_y,blockDim.x), iDivUp(diameter_y,blockDim.y));
	const dim3 gridDim_uv(iDivUp(ceilf(radius),blockDim.x), iDivUp(ceilf(radius),blockDim.y));

	uint8_t y = static_cast<uint8_t>(((int)(30 * color.x) + (int)(59 * color.y) + (int)(11 * color.z)) / 100);
	uint8_t u = static_cast<uint8_t>(((int)(-17 * color.x) - (int)(33 * color.y) + (int)(50 * color.z) + 12800) / 100);
	uint8_t v = static_cast<uint8_t>(((int)(50 * color.x) - (int)(42 * color.y) - (int)(8 * color.z) + 12800) / 100);

	gpuDrawCircleYY<<<gridDim,blockDim>>>((unsigned char *)input, width, height, offset_x, offset_y, cx, cy, radius*radius, y);
	// gpuDrawCircleUV<<<gridDim,gridDim_uv>>>((unsigned char *)input, width, height/2, offset_x/2, offset_y/2, cx/2, cy/2, radius*radius/4, u, v);


	return hipGetLastError();
}

hipError_t cudaDrawCircleOnYUV420( void* input_y, void* input_u,void* input_v, size_t width, size_t height, imageFormat format, int cx, int cy, float radius, const float4& color )
{
	// this is my function to draw cirle on Y channel of YUV image.. 
	if( !input_y || !input_u || !input_v  || width == 0 || height == 0 || radius <= 0 )
		return hipErrorInvalidValue;

	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	// if( input != output )
	// 	CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the circle
	const int diameter_y = ceilf(radius * 2.0f);
	const int offset_x = cx - radius;
	const int offset_y = cy - radius;
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim_y(iDivUp(diameter_y,blockDim.x), iDivUp(diameter_y,blockDim.y));
	const dim3 gridDim_uv(iDivUp(ceilf(radius),blockDim.x), iDivUp(ceilf(radius),blockDim.y));

	uint8_t y = static_cast<uint8_t>(((int)(30 * color.x) + (int)(59 * color.y) + (int)(11 * color.z)) / 100);
	uint8_t u = static_cast<uint8_t>(((int)(-17 * color.x) - (int)(33 * color.y) + (int)(50 * color.z) + 12800) / 100);
	uint8_t v = static_cast<uint8_t>(((int)(50 * color.x) - (int)(42 * color.y) - (int)(8 * color.z) + 12800) / 100);

	gpuDrawCircleOnOnePlane<uchar><<<gridDim_y,blockDim>>>((uchar*)input_y, width, height, offset_x, offset_y, cx, cy, radius*radius, y, 2048);
	gpuDrawCircleOnOnePlane<uchar><<<gridDim_uv,blockDim>>>((uchar*)input_u, width/2, height/2, offset_x/2, offset_y/2, cx/2, cy/2, radius*radius/4, u,1024);
	gpuDrawCircleOnOnePlane<uchar><<<gridDim_uv,blockDim>>>((uchar*)input_v, width/2, height/2, offset_x/2, offset_y/2, cx/2, cy/2, radius*radius/4, v, 1024);

	return hipGetLastError();
}



//----------------------------------------------------------------------------
// Line drawing (find if the distance to the line <= line_width)
// Distance from point to line segment - https://stackoverflow.com/a/1501725
//----------------------------------------------------------------------------
inline __device__ float lineDistanceSquared(float x, float y, float x1, float y1, float x2, float y2) 
{
	const float d = dist2(x1, y1, x2, y2);
	const float t = ((x-x1) * (x2-x1) + (y-y1) * (y2-y1)) / d;
	const float u = MAX(0, MIN(1, t));
	
	return dist2(x, y, x1 + u * (x2 - x1), y1 + u * (y2 - y1));
}
				 
template<typename T>
__global__ void gpuDrawLine( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int x1, int y1, int x2, int y2, const float4 color, float line_width2 ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	if( lineDistanceSquared(x, y, x1, y1, x2, y2) <= line_width2 )
	{
		const int idx = y * imgWidth + x;
		img[idx] = cudaAlphaBlend(img[idx], color);
	}
}

// cudaDrawLine
hipError_t cudaDrawLine( void* input, void* output, size_t width, size_t height, imageFormat format, int x1, int y1, int x2, int y2, const float4& color, float line_width )
{
	if( !input || !output || width == 0 || height == 0 || line_width <= 0 )
		return hipErrorInvalidValue;
	
	// check for lines < 2 pixels in length
	if( dist(x1,y1,x2,y2) < 2.0 )
	{
		LogWarning(LOG_CUDA "cudaDrawLine() - line has length < 2, skipping (%i,%i) (%i,%i)\n", x1, y1, x2, y2);
		return hipSuccess;
	}
	
	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the line
	const int left = MIN(x1,x2) - line_width;
	const int right = MAX(x1,x2) + line_width;
	const int top = MIN(y1,y2) - line_width;
	const int bottom = MAX(y1,y2) + line_width;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(right - left, blockDim.x), iDivUp(bottom - top, blockDim.y));

	#define LAUNCH_DRAW_LINE(type) \
		gpuDrawLine<type><<<gridDim, blockDim>>>((type*)output, width, height, left, top, x1, y1, x2, y2, color, line_width * line_width)
	
	if( format == IMAGE_RGB8 )
		LAUNCH_DRAW_LINE(uchar3);
	else if( format == IMAGE_RGBA8 )
		LAUNCH_DRAW_LINE(uchar4);
	else if( format == IMAGE_RGB32F )
		LAUNCH_DRAW_LINE(float3); 
	else if( format == IMAGE_RGBA32F )
		LAUNCH_DRAW_LINE(float4);
	else
	{
		imageFormatErrorMsg(LOG_CUDA, "cudaDrawLine()", format);
		return hipErrorInvalidValue;
	}
		
	return hipGetLastError();
}



//----------------------------------------------------------------------------
// Rect drawing (a grid of threads is launched over the rect)
//----------------------------------------------------------------------------
template<typename T>
__global__ void gpuDrawRect( T* img, int imgWidth, int imgHeight, int x0, int y0, int boxWidth, int boxHeight, const float4 color ) 
{
	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( box_x >= boxWidth || box_y >= boxHeight )
		return;

	const int x = box_x + x0;
	const int y = box_y + y0;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int idx = y * imgWidth + x;
	img[idx] = cudaAlphaBlend(img[idx], color);
}


__global__ void gpuAlongSide( char* input_0, char* input_1, char* output, size_t width, size_t height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	// if( box_x >= boxWidth || box_y >= boxHeight )
	// 	return;

	// const int x  = box_x + x0;
	// const int y =box_y + y0;

	// if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
	// 	return;

	const int idx = y * 2048 + x;

	if(x<960)
		output[idx] = input_0[y*2048+x];
	else 
		output[idx] = input_1[y*2048+x];
	// img[idx] = cudaAlphaBlend(img[idx], color);
}


hipError_t cudaAlongSide( void* input_0, void* input_1, void* output, size_t width, size_t height)
{
		const dim3 blockDim(8, 8);
		const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

		gpuAlongSide<<<gridDim, blockDim>>>((char *) input_0,(char *) input_1, (char *) output,width,height);

		return hipGetLastError();
}


// cudaDrawRect
hipError_t cudaDrawRect( void* input, void* output, size_t width, size_t height, imageFormat format, int left, int top, int right, int bottom, const float4& color, const float4& line_color, float line_width )
{
	if( !input || !output || width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// make sure the coordinates are ordered
	if( left > right )
	{
		const int swap = left;
		left = right;
		right = swap;
	}
	
	if( top > bottom )
	{
		const int swap = top;
		top = bottom;
		bottom = swap;
	}
	
	const int boxWidth = right - left;
	const int boxHeight = bottom - top;
	
	if( boxWidth <= 0 || boxHeight <= 0 )
	{
		LogError(LOG_CUDA "cudaDrawRect() -- rect had width/height <= 0  left=%i top=%i right=%i bottom=%i\n", left, top, right, bottom);
		return hipErrorInvalidValue;
	}

	// rect fill
	if( color.w > 0 )
	{
		const dim3 blockDim(8, 8);
		const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));
				
		#define LAUNCH_DRAW_RECT(type) \
			gpuDrawRect<type><<<gridDim, blockDim>>>((type*)output, width, height, left, top, boxWidth, boxHeight, color)
		
		if( format == IMAGE_RGB8 )
			LAUNCH_DRAW_RECT(uchar3);
		else if( format == IMAGE_RGBA8 )
			LAUNCH_DRAW_RECT(uchar4);
		else if( format == IMAGE_RGB32F )
			LAUNCH_DRAW_RECT(float3); 
		else if( format == IMAGE_RGBA32F )
			LAUNCH_DRAW_RECT(float4);
		else
		{
			imageFormatErrorMsg(LOG_CUDA, "cudaDrawRect()", format);
			return hipErrorInvalidValue;
		}
	}
	
	// rect outline
	if( line_color.w > 0 && line_width > 0 )
	{
		int lines[4][4] = {
			{left, top, right, top},
			{right, top, right, bottom},
			{right, bottom, left, bottom},
			{left, bottom, left, top}
		};
		
		for( uint32_t n=0; n < 4; n++ )
			CUDA(cudaDrawLine(output, width, height, format, lines[n][0], lines[n][1], lines[n][2], lines[n][3], line_color, line_width));
	}
	
	return hipGetLastError();
}